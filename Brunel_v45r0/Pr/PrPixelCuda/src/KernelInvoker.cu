#include "KernelInvoker.cuh"
#include "Kernel.cuh"

#include <iostream>

extern int*   h_no_sensors;
extern int*   h_no_hits;
extern int*   h_sensor_Zs;
extern int*   h_sensor_hitStarts;
extern int*   h_sensor_hitNums;
extern int*   h_hit_IDs;
extern float* h_hit_Xs;
extern float* h_hit_Ys;
extern int*   h_hit_Zs;

hipError_t invokeParallelSearch(
    dim3                         numBlocks,
    dim3                         numThreads,
    const std::vector<uint8_t> & input,
    std::vector<uint8_t>       & solution,
    std::ostream               & logger) {
  // For now, just perform what we did before
  // (backwards compatibility)
  int* h_track_indexes;
  int* num_tracks;
  Track* tracks;

  logger << "Input pointer: " 
    << std::hex << "0x" << (long long int) &(input[0])
    << std::dec << std::endl;

  setHPointersFromInput(const_cast<uint8_t*>(&input[0]), input.size());
  printInfo(logger);

  // int* h_prevs, *h_nexts;
  // Histo histo;

  char*  dev_input             = 0;
  int*   dev_num_tracks        = 0;
  int*   dev_track_indexes     = 0;
  Track* dev_tracks            = 0;
  bool*  dev_track_holders     = 0;
  int*   dev_prevs             = 0;
  int*   dev_nexts             = 0;
  int*   dev_tracks_to_process = 0;
  hipError_t cudaStatus = hipSuccess;

  // Choose which GPU to run on, change this on a multi-GPU system.
  cudaCheck(hipSetDevice(0));

  // Allocate memory
  // Allocate CPU buffers
  tracks = (Track*) malloc(MAX_TRACKS * sizeof(Track));
  //solution.resize(MAX_TRACKS * sizeof(Track));
  //tracks = (Track*) &(solution[0]);
  num_tracks = (int*) malloc(sizeof(int));

  int* h_prevs = (int*) malloc(h_no_hits[0] * sizeof(int));
  int* h_nexts = (int*) malloc(h_no_hits[0] * sizeof(int));
  bool* h_track_holders = (bool*) malloc(MAX_TRACKS * sizeof(bool));
  h_track_indexes = (int*) malloc(MAX_TRACKS * sizeof(int));

  // Allocate GPU buffers
  cudaCheck(hipMalloc((void**)&dev_tracks, MAX_TRACKS * sizeof(Track)));
  cudaCheck(hipMalloc((void**)&dev_track_holders, MAX_TRACKS * sizeof(bool)));
  cudaCheck(hipMalloc((void**)&dev_track_indexes, MAX_TRACKS * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_tracks_to_process, MAX_TRACKS * sizeof(int)));

  cudaCheck(hipMalloc((void**)&dev_prevs, h_no_hits[0] * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_nexts, h_no_hits[0] * sizeof(int)));

  // Copy input file from host memory to GPU buffers
  cudaCheck(hipMalloc((void**)&dev_input, input.size()));
  cudaCheck(hipMalloc((void**)&dev_num_tracks, sizeof(int)));

  // memcpys
  cudaCheck(hipMemcpy(dev_input, &(input[0]), input.size(), hipMemcpyHostToDevice));

  // Launch a kernel on the GPU with one thread for each element.
  prepareData<<<1, 1>>>(dev_input, dev_prevs, dev_nexts, dev_track_holders);

  // gpuKalman
  logger << "gpuKalman" << std::endl;
  hipEvent_t start_kalman, start_postprocess, stop;
  float t0, t1, t2;

  hipEventCreate(&start_kalman);
  hipEventCreate(&start_postprocess);
  hipEventCreate(&stop);

  hipEventRecord(start_kalman, 0 );

  gpuKalman<<<numBlocks, numThreads>>>(dev_tracks, dev_track_holders);

  hipEventRecord(start_postprocess);


  logger << "postProcess" << std::endl;
  postProcess<<<1, numThreads>>>(dev_tracks, dev_track_holders, dev_track_indexes, dev_num_tracks, dev_tracks_to_process);

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );

  hipEventElapsedTime( &t0, start_kalman, start_postprocess );
  hipEventElapsedTime( &t1, start_postprocess, stop );
  hipEventElapsedTime( &t2, start_kalman, stop );
  hipEventDestroy( start_kalman );
  hipEventDestroy( start_postprocess );
  hipEventDestroy( stop );

  // Get results
  cudaCheck(hipMemcpy(h_track_holders, dev_track_holders, MAX_TRACKS * sizeof(bool), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(h_track_indexes, dev_track_indexes, MAX_TRACKS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(tracks, dev_tracks, MAX_TRACKS * sizeof(Track), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(num_tracks, dev_num_tracks, sizeof(int), hipMemcpyDeviceToHost));

  // number of tracks after stage#1
  int no_tracks_stage1 = 0;
  for(int i=0; i<h_no_hits[0]; ++i)
    if(h_track_holders[i])
      ++no_tracks_stage1;

  // copy selected track to the solution vector

  if (*num_tracks > 0) {
    solution.resize(*num_tracks * sizeof(Track));
    Track * solutionTracks = (Track*)&solution[0];
    for (size_t i = 0; i != *num_tracks; ++i)
      solutionTracks[i] = tracks[h_track_indexes[i]];
  }

  // print debug info

  for(int i=0; i<num_tracks[0]; ++i)
    printTrack(tracks, h_track_indexes[i], logger);
  logger << "Processed " << num_tracks[0] << " tracks" << std::endl;

  free(h_prevs);
  free(h_nexts);
  free(h_track_holders);
  free(tracks);
  free(num_tracks);

  return cudaStatus;
}

// #track, h0, h1, h2, h3, ..., hn, length, chi2
void printTrack(Track* tracks, int track_no, std::ostream& logger){
  logger << track_no << ": ";

  Track t = tracks[track_no];
  for(int i=0; i<t.hitsNum; ++i){
    logger << h_hit_IDs[t.hits[i]] << ", ";
  }

  logger << "length: " << (int) t.hitsNum << std::endl;
}

void printOutAllSensorHits(int* prevs, int* nexts, std::ostream& logger){
  logger << "All valid sensor hits: " << std::endl;
  for(int i=0; i<h_no_sensors[0]; ++i){
    for(int j=0; j<h_sensor_hitNums[i]; ++j){
      int hit = h_sensor_hitStarts[i] + j;

      if(nexts[hit] != -1){
        std::cout << hit << ", " << nexts[hit] << std::endl;
      }
    }
  }
}

void printOutSensorHits(int sensorNumber, int* prevs, int* nexts, std::ostream& logger){
  for(int i=0; i<h_sensor_hitNums[sensorNumber]; ++i){
    int hstart = h_sensor_hitStarts[sensorNumber];

    logger << hstart + i << ": " << prevs[hstart + i] << ", " << nexts[hstart + i] << std::endl;
  }
}

void printInfo(std::ostream& logger) {
  logger << "Read info:" << std::endl
    << " no sensors: " << h_no_sensors[0] << std::endl
    << " no hits: " << h_no_hits[0] << std::endl
    << "First 5 sensors: " << std::endl;

  for (int i=0; i<5; ++i){
    logger << " Zs: " << h_sensor_Zs[i] << std::endl
      << " hitStarts: " << h_sensor_hitStarts[i] << std::endl
      << " hitNums: " << h_sensor_hitNums[i] << std::endl << std::endl;
  }

  logger << "First 5 hits: " << std::endl;

  for (int i=0; i<5; ++i){
    logger << " hit_id: " << h_hit_IDs[i] << std::endl
      << " hit_X: " << h_hit_Xs[i] << std::endl
      << " hit_Y: " << h_hit_Ys[i] << std::endl
      << " hit_Z: " << h_hit_Zs[i] << std::endl << std::endl;
  }
}

void getMaxNumberOfHits(char*& input, int& maxHits){
  int* l_no_sensors = (int*) &input[0];
  int* l_no_hits = (int*) (l_no_sensors + 1);
  int* l_sensor_Zs = (int*) (l_no_hits + 1);
  int* l_sensor_hitStarts = (int*) (l_sensor_Zs + l_no_sensors[0]);
  int* l_sensor_hitNums = (int*) (l_sensor_hitStarts + l_no_sensors[0]);

  maxHits = 0;
  for(int i=0; i<l_no_sensors[0]; ++i){
    if(l_sensor_hitNums[i] > maxHits)
      maxHits = l_sensor_hitNums[i];
  }
}
