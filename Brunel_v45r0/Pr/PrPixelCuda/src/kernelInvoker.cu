
#include "kernelInvoker.cuh"
#include "kernel.cuh"


extern int* h_no_sensors;
extern int* h_no_hits;
extern int* h_sensor_Zs;
extern int* h_sensor_hitStarts;
extern int* h_sensor_hitNums;
extern int* h_hit_IDs;
extern float* h_hit_Xs;
extern float* h_hit_Ys;
extern int* h_hit_Zs;

#define cudaCheck(stmt) do {										\
        hipError_t err = stmt;										\
        if (err != hipSuccess) {									\
            std::cerr << "Failed to run " << #stmt << std::endl;    \
            return err;										        \
        }															\
    } while(0)

// Helper function for using CUDA to add vectors in parallel.
hipError_t invokeParallelSearch(dim3 numBlocks, dim3 numThreads,
	char* input, int size, Track*& tracks, int*& num_tracks, int*& h_track_indexes){
    
	// int* h_prevs, *h_nexts;
	// Histo histo;

	char *dev_input = 0;
	int* dev_num_tracks = 0;
	int* dev_track_indexes = 0;
	Track *dev_tracks = 0;
	bool* dev_track_holders = 0;
	int* dev_prevs = 0;
	int* dev_nexts = 0;
	int* dev_tracks_to_process = 0;
    hipError_t cudaStatus = hipSuccess;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaCheck(hipSetDevice(0));
    
	// Allocate memory
	// Allocate CPU buffers
	tracks = (Track*) malloc(MAX_TRACKS * sizeof(Track));
	num_tracks = (int*) malloc(sizeof(int));

	int* h_prevs = (int*) malloc(h_no_hits[0] * sizeof(int));
	int* h_nexts = (int*) malloc(h_no_hits[0] * sizeof(int));
	bool* h_track_holders = (bool*) malloc(MAX_TRACKS * sizeof(bool));
	h_track_indexes = (int*) malloc(MAX_TRACKS * sizeof(int));

    // Allocate GPU buffers
    cudaCheck(hipMalloc((void**)&dev_tracks, MAX_TRACKS * sizeof(Track)));
	cudaCheck(hipMalloc((void**)&dev_track_holders, MAX_TRACKS * sizeof(bool)));
	cudaCheck(hipMalloc((void**)&dev_track_indexes, MAX_TRACKS * sizeof(int)));
	cudaCheck(hipMalloc((void**)&dev_tracks_to_process, MAX_TRACKS * sizeof(int)));

	cudaCheck(hipMalloc((void**)&dev_prevs, h_no_hits[0] * sizeof(int)));
	cudaCheck(hipMalloc((void**)&dev_nexts, h_no_hits[0] * sizeof(int)));
    
    // Copy input file from host memory to GPU buffers
    cudaCheck(hipMalloc((void**)&dev_input, size));
    cudaCheck(hipMalloc((void**)&dev_num_tracks, sizeof(int)));
    
	// memcpys
    cudaCheck(hipMemcpy(dev_input, input, size, hipMemcpyHostToDevice));

	// Launch a kernel on the GPU with one thread for each element.
	prepareData<<<1, 1>>>(dev_input, dev_prevs, dev_nexts, dev_track_holders);

	// gpuKalman
	std::cout << "gpuKalman" << std::endl;
	hipEvent_t start_kalman, start_postprocess, stop;
	float t0, t1, t2;

	hipEventCreate(&start_kalman);
	hipEventCreate(&start_postprocess);
	hipEventCreate(&stop);

	hipEventRecord(start_kalman, 0 );

	gpuKalman<<<46, 32>>>(dev_tracks, dev_track_holders);

	hipEventRecord(start_postprocess);
	
	/*cudaCheck(hipMemcpy(h_track_holders, dev_track_holders, MAX_TRACKS * sizeof(bool), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(tracks, dev_tracks, MAX_TRACKS * sizeof(Track), hipMemcpyDeviceToHost));
	int j = 0;
	for(int i=0; i<h_no_hits[0]; ++i){
		if(h_track_holders[i]){
			++j;
	// 		printTrack(tracks, i);
		}
	}
	std::cout <<  "gpuKalman: " << j << " tracks" << std::endl;
	// histo.plotChi2("after-kalman.root", h_track_holders, tracks, h_no_hits[0]);
	*/

	std::cout << "postProcess" << std::endl;
	postProcess<<<1, 32>>>(dev_tracks, dev_track_holders, dev_track_indexes, dev_num_tracks, dev_tracks_to_process);
	
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	hipEventElapsedTime( &t0, start_kalman, start_postprocess );
	hipEventElapsedTime( &t1, start_postprocess, stop );
	hipEventElapsedTime( &t2, start_kalman, stop );
	hipEventDestroy( start_kalman );
	hipEventDestroy( start_postprocess );
	hipEventDestroy( stop );

	// Get results
    cudaCheck(hipMemcpy(h_track_holders, dev_track_holders, MAX_TRACKS * sizeof(bool), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(h_track_indexes, dev_track_indexes, MAX_TRACKS * sizeof(int), hipMemcpyDeviceToHost));
	cudaCheck(hipMemcpy(tracks, dev_tracks, MAX_TRACKS * sizeof(Track), hipMemcpyDeviceToHost));
    cudaCheck(hipMemcpy(num_tracks, dev_num_tracks, sizeof(int), hipMemcpyDeviceToHost));
	
	// Amount of tracks after stage#1
	int no_tracks_stage1 = 0;
	for(int i=0; i<h_no_hits[0]; ++i)
               if(h_track_holders[i])
                       ++no_tracks_stage1;

	// Append to results
	// std::ofstream ofs;
	// ofs.open ("results_gpu.csv", std::ofstream::out | std::ofstream::app);
	// ofs << toString(h_no_hits[0]) << ", " << toString(no_tracks_stage1) << ", " << toString(num_tracks[0]) << ", " << toString(t0) << ", " << toString(t1) << ", " << toString(t2) << "\n";
	// ofs.close();

	// Produce some valuable statistics
        // Append to results
        //std::ofstream statf;
        //statf.open ("hits_per_sensor.csv", std::ofstream::out | std::ofstream::app);
        //statf << h_no_hits[0];
	//for(int i=0; i<48; ++i)
	//	statf << ", " << h_sensor_hitNums[i];
	//statf << "\n";
        //statf.close();


	for(int i=0; i<num_tracks[0]; ++i){
		printTrack(tracks, h_track_indexes[i]);
	}
	// std::cout << "postProcess: " << num_tracks[0] << " tracks" << std::endl << std::endl;
	std::cout << "Processed " << num_tracks[0] << " tracks" << std::endl;

	free(h_prevs);
	free(h_nexts);
	free(h_track_holders);

	// histo.plotChi2("after-post-processing.root", h_track_indexes, tracks, num_tracks[0]);

    // neighboursFinder<<<numBlocks, numThreads>>>();

	// Visualize results
	// cudaCheck(hipMemcpy(h_prevs, dev_prevs, h_no_hits[0] * sizeof(int), hipMemcpyDeviceToHost));
	// cudaCheck(hipMemcpy(h_nexts, dev_nexts, h_no_hits[0] * sizeof(int), hipMemcpyDeviceToHost));
	// printOutSensorHits(2, h_prevs, h_nexts);

	/*
	out = std::ofstream("prevnexts.out");
	out.write((char*) &h_prevs[0], h_no_hits[0] * sizeof(int));
	out.write((char*) &h_nexts[0], h_no_hits[0] * sizeof(int));
	out.close();
	*/

	// neighboursCleaner<<<numBlocks, numThreads>>>();
	
	// Visualize results
	// cudaCheck(hipMemcpy(h_prevs, dev_prevs, h_no_hits[0] * sizeof(int), hipMemcpyDeviceToHost));
	// cudaCheck(hipMemcpy(h_nexts, dev_nexts, h_no_hits[0] * sizeof(int), hipMemcpyDeviceToHost));
	// printOutSensorHits(2, h_prevs, h_nexts);
	// printOutAllSensorHits(h_prevs, h_nexts);
	
	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
 //    cudaCheck(hipDeviceSynchronize());
    
	// // cuda copy back
	// cudaCheck(hipMemcpy(num_tracks, dev_num_tracks, sizeof(int), hipMemcpyDeviceToHost));
	
 //    // Copy output vector from GPU buffer to host memory.
 //    cudaCheck(hipMemcpy(tracks, dev_tracks, num_tracks[0] * sizeof(Track), hipMemcpyDeviceToHost));
    
    return cudaStatus;
}

// #track, h0, h1, h2, h3, ..., hn, length, chi2
void printTrack(Track* tracks, int track_no){
	std::cout << track_no << ": ";

	Track t = tracks[track_no];
	for(int i=0; i<t.hitsNum; ++i){
		std::cout << h_hit_IDs[t.hits[i]] << ", ";
	}

	std::cout << "length: " << (int) t.hitsNum << std::endl;
}

/*
float f_chi2(Track& t)
{
	float ch = 0.0;
	int nDoF  = -4;
	int hitNumber;
	for (int i=0; i<t.hitsNum; ++i){
		hitNumber = t.hits[i];
		ch += f_chi2Track(t, hitNumber);
		nDoF += 2;
	}
	return ch/nDoF;
}
*/

void printOutAllSensorHits(int* prevs, int* nexts){
	std::cout << "All valid sensor hits: " << std::endl;
	for(int i=0; i<h_no_sensors[0]; ++i){
		for(int j=0; j<h_sensor_hitNums[i]; ++j){
			int hit = h_sensor_hitStarts[i] + j;
			
			if(nexts[hit] != -1){
				std::cout << hit << ", " << nexts[hit] << std::endl;
			}
		}
	}
}

void printOutSensorHits(int sensorNumber, int* prevs, int* nexts){
	for(int i=0; i<h_sensor_hitNums[sensorNumber]; ++i){
		int hstart = h_sensor_hitStarts[sensorNumber];

		std::cout << hstart + i << ": " << prevs[hstart + i] << ", " << nexts[hstart + i] << std::endl;
	}
}

void getMaxNumberOfHits(char*& input, int& maxHits){
	int* l_no_sensors = (int*) &input[0];
    int* l_no_hits = (int*) (l_no_sensors + 1);
    int* l_sensor_Zs = (int*) (l_no_hits + 1);
    int* l_sensor_hitStarts = (int*) (l_sensor_Zs + l_no_sensors[0]);
    int* l_sensor_hitNums = (int*) (l_sensor_hitStarts + l_no_sensors[0]);

	maxHits = 0;
	for(int i=0; i<l_no_sensors[0]; ++i){
		if(l_sensor_hitNums[i] > maxHits)
			maxHits = l_sensor_hitNums[i];
	}
}
