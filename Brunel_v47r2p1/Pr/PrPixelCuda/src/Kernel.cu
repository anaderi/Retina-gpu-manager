#include "hip/hip_runtime.h"
#include "Kernel.cuh"

#include <cstdio>

#define HITS_SHARED 32
#define MAX_FLOAT 100000000.0
#define POST_PROCESSING 0

__device__ int   *no_sensors;
__device__ int   *no_hits;
__device__ int   *sensor_Zs;
__device__ int   *sensor_hitStarts;
__device__ int   *sensor_hitNums;
__device__ int   *hit_IDs;
__device__ float *hit_Xs;
__device__ float *hit_Ys;
__device__ int   *hit_Zs;

__device__ int* prevs;
__device__ int* nexts;

__global__ void prepareData(char* input, int* _prevs, int* _nexts) {
  no_sensors       = (int*)   (input + 0);
  no_hits          = (int*)   (input + 4);
  sensor_Zs        = (int*)   (input + 8);
  sensor_hitStarts = (int*)   (sensor_Zs        + *no_sensors);
  sensor_hitNums   = (int*)   (sensor_hitStarts + *no_sensors);
  hit_IDs          = (int*)   (sensor_hitNums   + *no_sensors);
  hit_Xs           = (float*) (hit_IDs + *no_hits);
  hit_Ys           = (float*) (hit_Xs  + *no_hits);
  hit_Zs           = (int*)   (hit_Ys  + *no_hits);

  prevs = _prevs;
  nexts = _nexts;
}

/** fitHits, gives the fit between h0 and h1.

The accept condition requires dxmax and dymax to be in a range.

The fit (d1) depends on the distance of the tracklet to <0,0,0>.
*/
__device__ float fitHits(Hit& h0, Hit& h1, Hit &h2, Sensor& s0, Sensor& s1, Sensor& s2) {
  // Max dx, dy permissible over next hit

  // TODO: This can go outside this function (only calc once per pair
  // of sensors). Also, it could only be calculated on best fitting distance d1.
  float s_dist = fabs((float)(s1.z - s0.z));
  float dxmax = PARAM_MAXXSLOPE * s_dist;
  float dymax = PARAM_MAXYSLOPE * s_dist;

  bool accept_condition = fabs(h1.x - h0.x) < dxmax &&
              fabs(h1.y - h0.y) < dymax;

  /*float dxmax = PARAM_MAXXSLOPE * fabs((float)(s1.z - s0.z));
  float dymax = PARAM_MAXYSLOPE * fabs((float)(s1.z - s0.z));*/

  // Distance to <0,0,0> in its XY plane.
  /*float t = - s0.z / (s1.z - s0.z);
  float x = h0.x + t * (h1.x - h0.x);
  float y = h0.y + t * (h1.y - h0.y);
  float d1 = sqrtf(powf( (float) (x), 2.0f) +
        powf((float) (y), 2.0f));*/

  // Distance between the hits.
  // float d1 = sqrtf(powf( (float) (h1.x - h0.x), 2.0f) +
  //          powf((float) (h1.y - h0.y), 2.0f));

  // Distance between line <h0,h1> and h2 in XY plane (s2.z)
  // float t = s2.z - s0.z / (s1.z - s0.z);
  // float x = h0.x + t * (h1.x - h0.x);
  // float y = h0.y + t * (h1.y - h0.y);
  // float d1 = sqrtf(powf( (float) (x - h2.x), 2.0f) +
  //      powf((float) (y - h2.y), 2.0f));
  // accept_condition &= (fabs(x - h2.x) < PARAM_TOLERANCE);

  // Require chi2 of third hit below the threshold
  // float t = ((float) (s2.z - s0.z)) / ((float) (s1.z - s0.z));
  float z2_tz = ((float) s2.z - s0.z) / ((float) (s1.z - s0.z));
  float x = h0.x + (h1.x - h0.x) * z2_tz;
  float y = h0.y + (h1.y - h0.y) * z2_tz;

  float dx = x - h2.x;
  float dy = y - h2.y;
  float chi2 = dx * dx * PARAM_W + dy * dy * PARAM_W;
  accept_condition &= chi2 < PARAM_MAXCHI2;

  return accept_condition * chi2 + !accept_condition * MAX_FLOAT;
}

// TODO: Optimize with Olivier's
__device__ float fitHitToTrack(Track& t, Hit& h1, Sensor& s1) {
  // tolerance
  // TODO: To improve efficiency, try with PARAM_TOLERANCE_EXTENDED
  float x_prediction = t.x0 + t.tx * s1.z;
  bool tol_condition = fabs(x_prediction - h1.x) < PARAM_TOLERANCE;

  // chi2 of hit (taken out from function for efficiency)
  float dx = x_prediction - h1.x;
  float dy = (t.y0 + t.ty * s1.z) - h1.y;
  float chi2 = dx * dx * PARAM_W + dy * dy * PARAM_W;

  // TODO: The check for chi2_condition can totally be done after this call
  bool chi2_condition = chi2 < PARAM_MAXCHI2;

  return tol_condition * chi2_condition * chi2 + (!tol_condition || !chi2_condition) * MAX_FLOAT;
}

// Create track
__device__ void acceptTrack(Track& t, TrackFit& fit, Hit& h0, Hit& h1, Sensor& s0, Sensor& s1, int h0_num, int h1_num) {
  float wz = PARAM_W * s0.z;

  fit.s0 = PARAM_W;
  fit.sx = PARAM_W * h0.x;
  fit.sz = wz;
  fit.sxz = wz * h0.x;
  fit.sz2 = wz * s0.z;

  fit.u0 = PARAM_W;
  fit.uy = PARAM_W * h0.y;
  fit.uz = wz;
  fit.uyz = wz * h0.y;
  fit.uz2 = wz * s0.z;

  t.hitsNum = 1;
  t.hits[0] = h0_num;

  // note: This could be done here (inlined)
  updateTrack(t, fit, h1, s1, h1_num);
}

// Update track
__device__ void updateTrack(Track& t, TrackFit& fit, Hit& h1, Sensor& s1, int h1_num) {
  float wz = PARAM_W * s1.z;

  fit.s0 += PARAM_W;
  fit.sx += PARAM_W * h1.x;
  fit.sz += wz;
  fit.sxz += wz * h1.x;
  fit.sz2 += wz * s1.z;

  fit.u0 += PARAM_W;
  fit.uy += PARAM_W * h1.y;
  fit.uz += wz;
  fit.uyz += wz * h1.y;
  fit.uz2 += wz * s1.z;

  t.hits[t.hitsNum] = h1_num;
  t.hitsNum++;

  updateTrackCoords(t, fit);
}

// TODO: Check this function
__device__ void updateTrackCoords (Track& t, TrackFit& fit) {
  float den = (fit.sz2 * fit.s0 - fit.sz * fit.sz);
  if (fabs(den) < 10e-10)
    den = 1.f;
  t.tx = (fit.sxz * fit.s0  - fit.sx  * fit.sz) / den;
  t.x0 = (fit.sx  * fit.sz2 - fit.sxz * fit.sz) / den;

  den = (fit.uz2 * fit.u0 - fit.uz * fit.uz);
  if (fabs(den) < 10e-10)
    den = 1.f;
  t.ty = (fit.uyz * fit.u0  - fit.uy  * fit.uz) / den;
  t.y0 = (fit.uy  * fit.uz2 - fit.uyz * fit.uz) / den;
}

/** Simple implementation of the Kalman Filter selection on the GPU (step 4).

Will rely on pre-processing for selecting next-hits for each hit.

Implementation,
- Perform implementation searching on all hits for each sensor

The algorithm has two parts:
- Track creation (two hits)
- Track following (consecutive sensors)


Optimizations,
- Optimize with shared memory
- Optimize further with pre-processing

Then there must be a post-processing, which selects the
best tracks based on (as per the conversation with David):
- length
- chi2

For this, simply use the table with all created tracks (postProcess):

#track, h0, h1, h2, h3, ..., hn, length, chi2

*/

__global__ void gpuKalman(Track* tracks, bool* trackHolders) {
  Track t;
  TrackFit tfit;
  Sensor s0, s1, s2;
  Hit h0, h1, h2;

  float fit, best_fit;
  bool fit_is_better, accept_track;
  int best_hit, best_hit_h2;

  // 4 of the sensors are unused, because the algorithm needs 5-8-sensor spans
  const int firstSensor  = (gridDim.x - blockIdx.x - 1 + 4);
  const int secondSensor = firstSensor - 2;
  const int lastSensor   = max(firstSensor - 7, 0);

  s0.hitStart = sensor_hitStarts [firstSensor];
  s0.hitNums  = sensor_hitNums   [firstSensor];
  s0.z        = sensor_Zs        [firstSensor];

  s1.hitStart = sensor_hitStarts [secondSensor];
  s1.hitNums  = sensor_hitNums   [secondSensor];
  s1.z        = sensor_Zs        [secondSensor];

  // Iterate over all hits for the current sensor
  const int hits_per_thread = (s0.hitNums + blockDim.x - 1) / blockDim.x;
  for (int i = 0; i < hits_per_thread; ++i) {
    const int current_hit = threadIdx.x * hits_per_thread + i;
    if (current_hit < s0.hitNums) {
      h0.x = hit_Xs[s0.hitStart + current_hit];
      h0.y = hit_Ys[s0.hitStart + current_hit];

      // Initialize track
      for (int j = 0; j < MAX_TRACK_SIZE; ++j)
        t.hits[j] = -1;

      // TRACK CREATION
      // TODO: Modify with preprocessed list of hits.
      best_fit    = MAX_FLOAT;
      best_hit    = -1;
      best_hit_h2 = -1;
      for (int j = 0; j < sensor_hitNums[secondSensor]; ++j) {
        // TODO: Load in chunks of SHARED_MEMORY and take
        // them from shared memory.
        h1.x = hit_Xs[s1.hitStart + j];
        h1.y = hit_Ys[s1.hitStart + j];

        // Search in both directions
        for (int thirdSensor = firstSensor - 3; thirdSensor != lastSensor; --thirdSensor) {
          // TODO: shared memory.
          s2.hitStart = sensor_hitStarts [thirdSensor];
          s2.hitNums  = sensor_hitNums   [thirdSensor];
          s2.z        = sensor_Zs        [thirdSensor];

          // Iterate in the third! list of hits
          for (int k = 0; k < sensor_hitNums[thirdSensor]; ++k) {
            // TODO: Load in chunks of SHARED_MEMORY and take
            // them from shared memory.
            h2.x = hit_Xs[s2.hitStart + k];
            h2.y = hit_Ys[s2.hitStart + k];

            fit = fitHits(h0, h1, h2, s0, s1, s2);
            fit_is_better = fit < best_fit;

            best_fit    = fit_is_better * fit + !fit_is_better * best_fit;
            best_hit    = fit_is_better * j   + !fit_is_better * best_hit;
            best_hit_h2 = fit_is_better * k   + !fit_is_better * best_hit_h2;
          }
        }
      }

      accept_track = best_fit != MAX_FLOAT;

      // We have a best fit!

      // For those that have tracks, we go on
      if (accept_track) {
        // Fill in t (ONLY in case the best fit is acceptable)
        acceptTrack(t, tfit, h0, h1, s0, s1, s0.hitStart + current_hit, s1.hitStart + best_hit);
        updateTrack(t, tfit, h2, s2, s2.hitStart + best_hit_h2);

        // TRACK FOLLOWING
        for (int followSensor = secondSensor - 4; followSensor >= 0; followSensor -= 2) {
          s1.hitStart = sensor_hitStarts[followSensor];
          s1.hitNums  = sensor_hitNums[followSensor];
          s1.z        = sensor_Zs[followSensor];

          best_fit = MAX_FLOAT;
          for (int k = 0; k < sensor_hitNums[followSensor]; ++k) {
            // TODO: Load in chunks of SHARED_MEMORY and take them from shared memory.
            h1.x = hit_Xs[s1.hitStart + k];
            h1.y = hit_Ys[s1.hitStart + k];

            fit = fitHitToTrack(t, h1, s1);
            fit_is_better = fit < best_fit;

            best_fit = fit_is_better * fit + !fit_is_better * best_fit;
            best_hit = fit_is_better * k   + !fit_is_better * best_hit;
          }

          // We have a best fit!
          // Fill in t, ONLY in case the best fit is acceptable

          // TODO: Maybe try to do this more "parallel"
          if (best_fit != MAX_FLOAT)
            updateTrack(t, tfit, h1, s1, s1.hitStart + best_hit);
        }
      }

      // If it's a track, write it to memory, no matter what kind
      // of track it is.
      trackHolders[s0.hitStart + current_hit] = accept_track && (t.hitsNum >= MIN_HITS_TRACK);
      if (accept_track && (t.hitsNum >= MIN_HITS_TRACK))
        tracks[s0.hitStart + current_hit] = t;
    }
  }
}

__global__ void gpuKalmanBalanced(Span  * spans, Fit * fittings) {
  Sensor s0, s1, s2;
  Hit    h0, h1, h2;

  Span span = spans[blockDim.x * blockIdx.x + threadIdx.x];

  const int firstSensor  = span.firstSensor;
  const int secondSensor = firstSensor - 2;
  const int lastSensor   = max(firstSensor - 7, 0);

	s0.z        = sensor_Zs        [firstSensor];
	s0.hitNums  = sensor_hitNums   [firstSensor];
	s0.hitStart = sensor_hitStarts [firstSensor];

	s1.z        = sensor_Zs        [secondSensor];
	s1.hitNums  = sensor_hitNums   [secondSensor];
	s1.hitStart = sensor_hitStarts [secondSensor];

  float bestFit    = MAX_FLOAT;
  int   bestHit0   = -1;
  int   bestHit1   = -1;
  int   bestHit2   = -1;
  int   bestSensor = -1;

	// iterate through a portion of candidate hits defined by
	// start and end coordinates in lexicographic order
  int i = span.first.s0;
  int j = span.first.s1;
  while ((i != span.last.s0 || j != span.last.s1) && i != s0.hitNums) {
    h0.x = hit_Xs[s0.hitStart + i];
    h0.y = hit_Ys[s0.hitStart + i];

    h1.x = hit_Xs[s1.hitStart + j];
    h1.y = hit_Ys[s1.hitStart + j];

    for (int thirdSensor = firstSensor - 3; thirdSensor != lastSensor; --thirdSensor) {
      s2.z        = sensor_Zs        [thirdSensor];
      s2.hitNums  = sensor_hitNums   [thirdSensor];
      s2.hitStart = sensor_hitStarts [thirdSensor];

      for (int k = 0; k != s2.hitNums; ++k) {
        h2.x = hit_Xs[s2.hitStart + k];
        h2.y = hit_Ys[s2.hitStart + k];

        float fit = fitHits(h0, h1, h2, s0, s1, s2);

        bool isBetter = fit < bestFit;

        bestFit    = isBetter * fit         + !isBetter * bestFit;
        bestHit0   = isBetter * i           + !isBetter * bestHit0;
        bestHit1   = isBetter * j           + !isBetter * bestHit1;
        bestHit2   = isBetter * k           + !isBetter * bestHit2;
        bestSensor = isBetter * thirdSensor + !isBetter * thirdSensor;
      }
    }
    // iterate in lexicographic order
    ++j;
    if (j == s1.hitNums) {
      j = 0;
      ++i;
    }
  }
  fittings[blockDim.x * blockIdx.x + threadIdx.x] =
    { bestFit, bestHit0, bestHit1, bestHit2, firstSensor, bestSensor };
}

__global__ void consolidateHits(Fit * fittings, int n, Track * tracks, bool * trackHolders) {
  if (n == 0)
    return;

  int sensor  = fittings[0].firstSensor;
  int bestFit = -1;

  for (int i = 0; i != n; ++i) {
    Fit fit = fittings[i];
    if (fit.firstSensor == sensor)
    {
      if (fit.fitness < bestFit)
        bestFit = i;
    }
    else
    {
      addTrack(fittings[bestFit], tracks, trackHolders);
      bestFit = i;
    }
  }
  //addTrack(fittings[bestFit], tracks, trackHolders);
}

__device__ void addTrack(const Fit & fit, Track * tracks, bool * trackHolders) {
  if (fit.fitness == MAX_FLOAT)
    return;

  int firstSensor  = fit.firstSensor;
  int secondSensor = firstSensor - 2;
  int thirdSensor  = fit.thirdSensor;

  Sensor s0 = { sensor_Zs[firstSensor],  sensor_hitStarts[firstSensor],  sensor_hitNums[firstSensor]  };
  Sensor s1 = { sensor_Zs[secondSensor], sensor_hitStarts[secondSensor], sensor_hitNums[secondSensor] };
  Sensor s2 = { sensor_Zs[thirdSensor],  sensor_hitStarts[thirdSensor],  sensor_hitNums[thirdSensor]  };

  Hit h0 = { hit_Xs[s0.hitStart + fit.hit0], hit_Ys[s0.hitStart + fit.hit0] };
  Hit h1 = { hit_Xs[s1.hitStart + fit.hit1], hit_Ys[s1.hitStart + fit.hit1] };
  Hit h2 = { hit_Xs[s2.hitStart + fit.hit2], hit_Ys[s2.hitStart + fit.hit2] };

  // Fill in t (ONLY in case the best fit is acceptable)
  Track    t;
  TrackFit tfit;
  acceptTrack(t, tfit, h0, h1, s0, s1, s0.hitStart + fit.hit0, s1.hitStart + fit.hit1);
  updateTrack(t, tfit, h2, s2, s2.hitStart + fit.hit2);

  // Track following
  for (int followSensor = firstSensor - 6; followSensor >= 0; followSensor -= 2) {
    s1.hitStart = sensor_hitStarts[followSensor];
    s1.hitNums  = sensor_hitNums[followSensor];
    s1.z        = sensor_Zs[followSensor];

    float bestFit = MAX_FLOAT;
    int   bestHit = 0;
    for (int k = 0; k < sensor_hitNums[followSensor]; ++k) {
      // TODO: Load in chunks of SHARED_MEMORY and take them from shared memory.
      h1.x = hit_Xs[s1.hitStart + k];
      h1.y = hit_Ys[s1.hitStart + k];

      float fitness = fitHitToTrack(t, h1, s1);
      bool isBetter = fitness < bestFit;
      bestFit = isBetter * fitness + !isBetter * bestFit;
      bestHit = isBetter * k       + !isBetter * bestHit;
    }

    // We have a best fit!
    // Fill in t, ONLY in case the best fit is acceptable

    // TODO: Maybe try to do this more "parallel"
    if (bestFit != MAX_FLOAT)
      updateTrack(t, tfit, h1, s1, s1.hitStart + bestHit);
  }

  // If it's a track, write it to memory, no matter what kind of track it is.
  if (t.hitsNum >= MIN_HITS_TRACK) {
    const int i = s0.hitStart + fit.hit0;
    tracks[i]       = t;
    trackHolders[i] = true;
  }
}


/* Calculating the chi2 of a track is quite cumbersome.
It implies loading hit_Xs, hit_Ys, and sensor_Zs elements for each
hit of the track. This introduces branching, and is slow.

However, the track chi2 has to be calculated only when the
track has been created (the tx, ty values change).
*/

__device__ float trackChi2(Track& t) {
  float ch = 0.0;
  int nDoF  = -4 + 2 * t.hitsNum;
  Hit h;
  for (int i = 0; i < MAX_TRACK_SIZE; i++) {
    // TODO: Maybe there's a better way to do this
    if (t.hits[i] != -1) {
      h.x = hit_Xs[t.hits[i]];
      h.y = hit_Ys[t.hits[i]];

      ch += hitChi2(t, h, hit_Zs[t.hits[i]]);
    }
  }
  return ch/nDoF;
}

__device__ float hitChi2(Track& t, Hit& h, int hit_z) {
  // chi2 of a hit
  float dx = (t.x0 + t.tx * hit_z) - h.x;
  float dy = (t.y0 + t.ty * hit_z) - h.y;
  return dx * dx * PARAM_W + dy * dy * PARAM_W;
}


/** The postProcess method takes care of discarding redundant tracks. In other
 words, it (hopefully) increases the purity of our tracks.

- Inspect trackHolders and generate track_indexes and num_tracks.

The main idea is to accept tracks which have unique (> REQUIRED_UNIQUES) hits.
For this, each track is checked against all other more preferent tracks, and non
common hits are kept.

TODO: Change the preference system into something more civilized.  A track t0
has preference over another t1 one if: t0.hitsNum > t1.hitsNum || (t0.hitsNum ==
t1.hitsNum && chi2(t0) < chi2(t1))
*/
__global__ void postProcess(Track* tracks, bool* trackHolders, int* track_indexes, int* num_tracks, int* tracks_to_process) {
  // tracks_to_process holds the list of tracks with trackHolders[t] == true

  // TODO: Try with sh_tracks_to_process
  // __shared__ int sh_tracks_to_process[MAX_POST_TRACKS];

  __shared__ Track sh_tracks [BUNCH_POST_TRACKS];
  __shared__ float sh_chi2   [BUNCH_POST_TRACKS];

  __shared__ Track sh_next_tracks [BUNCH_POST_TRACKS];
  __shared__ float sh_next_chi2   [BUNCH_POST_TRACKS];

  // We will use an atomic to write on a vector concurrently on several values
  __shared__ int tracks_to_process_size;
  __shared__ int tracks_accepted_size;

  tracks_to_process_size = 0;
  tracks_accepted_size   = 0;

  __syncthreads(); // for the atomics tracks_to_process_size, and tracks_processed

  int current_track, next_track;
  bool preferent;

  const int hitsPerBlock = (*no_hits + blockDim.x - 1) / blockDim.x;
  for (int i = 0; i < hitsPerBlock; ++i) {
    current_track = blockDim.x * i + threadIdx.x;
    if (current_track < *no_hits) {
      // Iterate in all tracks (current_track)

      if (trackHolders[current_track]) {
        // Atomic add
        int current_atomic = atomicAdd(&tracks_to_process_size, 1);

        // TODO: This condition shouldn't exist,
        // redo using method to process in batches if necessary
        // if (current_atomic < MAX_POST_TRACKS)
        tracks_to_process[current_atomic] = current_track;
      }
    }
  }

  __syncthreads();

  // Iterate in all current_tracks against all next_tracks.
  // Do this processing on batches of blockDim.x size
  const int tracksPerBlock = (tracks_to_process_size + blockDim.x - 1) / blockDim.x;
  for (int i = 0; i < tracksPerBlock; ++i) {
    current_track = blockDim.x * i + threadIdx.x;
    if (current_track < tracks_to_process_size) {
      // Store all tracks in sh_tracks
      sh_tracks[threadIdx.x] = tracks[tracks_to_process[current_track]];

      // Calculate chi2
      sh_chi2[threadIdx.x] = trackChi2(sh_tracks[threadIdx.x]);
    }

    __syncthreads();

    // Iterate in all next_tracks
    for (int j = 0; j < tracksPerBlock; ++j) {
      next_track = blockDim.x * j + threadIdx.x;

      if (next_track < tracks_to_process_size) {
        // Store all tracks in sh_tracks
        sh_next_tracks[threadIdx.x] = tracks[tracks_to_process[next_track]];

        // Calculate chi2
        sh_next_chi2[threadIdx.x] = trackChi2(sh_tracks[threadIdx.x]);
      }

      __syncthreads();

      // All is loaded, commencing assault!
      for (int k=0; k<BUNCH_POST_TRACKS; ++k) {
        next_track = blockDim.x * j + k;

        if (current_track < tracks_to_process_size && next_track < tracks_to_process_size) {
          /* Compare all tracks to check uniqueness, based on
          - length
          - chi2

          preferent is a boolean storing this logic. It reads,

          TODO: Change preference system by something more civilized
          next_track is preferent if
            it's not current_track,
            its length > current_track . length OR
            (its length == current_track . length AND
            chi2 < current_track . chi2)
          */
          preferent = current_track!=next_track &&
                    (sh_next_tracks[k].hitsNum > sh_tracks[threadIdx.x].hitsNum ||
                    (sh_next_tracks[k].hitsNum == sh_tracks[threadIdx.x].hitsNum &&
                    sh_next_chi2[k] < sh_chi2[threadIdx.x]));

          // Preference system based solely on chi2
          /*preferent = current_track!=next_track &&
                    sh_next_chi2[k] < sh_chi2[threadIdx.x]; */

          // TODO: Maybe there's a better way...
          if (preferent) {
            // Eliminate hits from current_track, based on next_track's
            for (int current_hit=0; current_hit<MAX_TRACK_SIZE; ++current_hit) {
              for (int next_hit=0; next_hit<MAX_TRACK_SIZE; ++next_hit) {
                /* apply mask:
                a[i] =
                  (a[i] == b[j]) * -1 +
                  (a[i] != b[j]) * a[i]
                */
                sh_tracks[threadIdx.x].hits[current_hit] =
                  (sh_tracks[threadIdx.x].hits[current_hit] == sh_next_tracks[k].hits[next_hit]) * -1 +
                  (sh_tracks[threadIdx.x].hits[current_hit] != sh_next_tracks[k].hits[next_hit]) *
                    sh_tracks[threadIdx.x].hits[current_hit];
              }
            }
          }
        }
      }
    }

    if (current_track < tracks_to_process_size) {
      // Check how many uniques do we have
      int unique = 0;
      for (int hit=0; hit<MAX_TRACK_SIZE; ++hit)
        unique += (sh_tracks[threadIdx.x].hits[hit]!=-1);

      if (!POST_PROCESSING || ((float) unique) / sh_tracks[threadIdx.x].hitsNum > REQUIRED_UNIQUES) {
        int current_track_accepted = atomicAdd(&tracks_accepted_size, 1);

        track_indexes[current_track_accepted] = tracks_to_process[current_track];
      }
    }
  }

  __syncthreads();

  if (threadIdx.x==0)
    *num_tracks = tracks_accepted_size;
}

