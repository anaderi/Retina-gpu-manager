#include "hip/hip_runtime.h"
#include "Kernel.cuh"

#include <cstdio>

#define HITS_SHARED 32
#define MAX_FLOAT 100000000.0
#define POST_PROCESSING 0

__device__ int* no_sensors;
__device__ int* no_hits;
__device__ int* sensor_Zs;
__device__ int* sensor_hitStarts;
__device__ int* sensor_hitNums;
__device__ int* hit_IDs;
__device__ float* hit_Xs;
__device__ float* hit_Ys;
__device__ int* hit_Zs;

__device__ int* prevs;
__device__ int* nexts;

__global__ void prepareData(char* input, int* _prevs, int* _nexts, bool* track_holders) {
  no_sensors       = (int*)   (input + 0);
  no_hits          = (int*)   (input + 4);
  sensor_Zs        = (int*)   (input + 8);
  sensor_hitStarts = (int*)   (sensor_Zs        + *no_sensors);
  sensor_hitNums   = (int*)   (sensor_hitStarts + *no_sensors);
  hit_IDs          = (int*)   (sensor_hitNums   + *no_sensors);
  hit_Xs           = (float*) (hit_IDs + *no_hits);
  hit_Ys           = (float*) (hit_Xs  + *no_hits);
  hit_Zs           = (int*)   (hit_Ys  + *no_hits);

  prevs = _prevs;
  nexts = _nexts;

  for (int i = 0; i < MAX_TRACKS; ++i)
    track_holders[i] = false;
}

/** fitHits, gives the fit between h0 and h1.

The accept condition requires dxmax and dymax to be in a range.

The fit (d1) depends on the distance of the tracklet to <0,0,0>.
*/
__device__ float fitHits(Hit& h0, Hit& h1, Hit &h2, Sensor& s0, Sensor& s1, Sensor& s2) {
  // Max dx, dy permissible over next hit

  // TODO: This can go outside this function (only calc once per pair
  // of sensors). Also, it could only be calculated on best fitting distance d1.
  float s_dist = fabs((float)(s1.z - s0.z));
  float dxmax = PARAM_MAXXSLOPE * s_dist;
  float dymax = PARAM_MAXYSLOPE * s_dist;

  bool accept_condition = fabs(h1.x - h0.x) < dxmax &&
              fabs(h1.y - h0.y) < dymax;

  /*float dxmax = PARAM_MAXXSLOPE * fabs((float)(s1.z - s0.z));
  float dymax = PARAM_MAXYSLOPE * fabs((float)(s1.z - s0.z));*/

  // Distance to <0,0,0> in its XY plane.
  /*float t = - s0.z / (s1.z - s0.z);
  float x = h0.x + t * (h1.x - h0.x);
  float y = h0.y + t * (h1.y - h0.y);
  float d1 = sqrtf(powf( (float) (x), 2.0f) +
        powf((float) (y), 2.0f));*/

  // Distance between the hits.
  // float d1 = sqrtf(powf( (float) (h1.x - h0.x), 2.0f) +
  //          powf((float) (h1.y - h0.y), 2.0f));

  // Distance between line <h0,h1> and h2 in XY plane (s2.z)
  // float t = s2.z - s0.z / (s1.z - s0.z);
  // float x = h0.x + t * (h1.x - h0.x);
  // float y = h0.y + t * (h1.y - h0.y);
  // float d1 = sqrtf(powf( (float) (x - h2.x), 2.0f) +
  //      powf((float) (y - h2.y), 2.0f));
  // accept_condition &= (fabs(x - h2.x) < PARAM_TOLERANCE);

  // Require chi2 of third hit below the threshold
  // float t = ((float) (s2.z - s0.z)) / ((float) (s1.z - s0.z));
  float z2_tz = ((float) s2.z - s0.z) / ((float) (s1.z - s0.z));
  float x = h0.x + (h1.x - h0.x) * z2_tz;
  float y = h0.y + (h1.y - h0.y) * z2_tz;

  float dx = x - h2.x;
  float dy = y - h2.y;
  float chi2 = dx * dx * PARAM_W + dy * dy * PARAM_W;
  accept_condition &= chi2 < PARAM_MAXCHI2;

  return accept_condition * chi2 + !accept_condition * MAX_FLOAT;
}

// TODO: Optimize with Olivier's
__device__ float fitHitToTrack(Track& t, Hit& h1, Sensor& s1) {
  // tolerance
  // TODO: To improve efficiency, try with PARAM_TOLERANCE_EXTENDED
  float x_prediction = t.x0 + t.tx * s1.z;
  bool tol_condition = fabs(x_prediction - h1.x) < PARAM_TOLERANCE;

  // chi2 of hit (taken out from function for efficiency)
  float dx = x_prediction - h1.x;
  float dy = (t.y0 + t.ty * s1.z) - h1.y;
  float chi2 = dx * dx * PARAM_W + dy * dy * PARAM_W;

  // TODO: The check for chi2_condition can totally be done after this call
  bool chi2_condition = chi2 < PARAM_MAXCHI2;

  return tol_condition * chi2_condition * chi2 + (!tol_condition || !chi2_condition) * MAX_FLOAT;
}

// Create track
__device__ void acceptTrack(Track& t, TrackFit& fit, Hit& h0, Hit& h1, Sensor& s0, Sensor& s1, int h0_num, int h1_num) {
  float wz = PARAM_W * s0.z;

  fit.s0 = PARAM_W;
  fit.sx = PARAM_W * h0.x;
  fit.sz = wz;
  fit.sxz = wz * h0.x;
  fit.sz2 = wz * s0.z;

  fit.u0 = PARAM_W;
  fit.uy = PARAM_W * h0.y;
  fit.uz = wz;
  fit.uyz = wz * h0.y;
  fit.uz2 = wz * s0.z;

  t.hitsNum = 1;
  t.hits[0] = h0_num;

  // note: This could be done here (inlined)
  updateTrack(t, fit, h1, s1, h1_num);
}

// Update track
__device__ void updateTrack(Track& t, TrackFit& fit, Hit& h1, Sensor& s1, int h1_num) {
  float wz = PARAM_W * s1.z;

  fit.s0 += PARAM_W;
  fit.sx += PARAM_W * h1.x;
  fit.sz += wz;
  fit.sxz += wz * h1.x;
  fit.sz2 += wz * s1.z;

  fit.u0 += PARAM_W;
  fit.uy += PARAM_W * h1.y;
  fit.uz += wz;
  fit.uyz += wz * h1.y;
  fit.uz2 += wz * s1.z;

  t.hits[t.hitsNum] = h1_num;
  t.hitsNum++;

  updateTrackCoords(t, fit);
}

// TODO: Check this function
__device__ void updateTrackCoords (Track& t, TrackFit& fit) {
  float den = (fit.sz2 * fit.s0 - fit.sz * fit.sz);
  if (fabs(den) < 10e-10)
    den = 1.f;
  t.tx = (fit.sxz * fit.s0  - fit.sx  * fit.sz) / den;
  t.x0 = (fit.sx  * fit.sz2 - fit.sxz * fit.sz) / den;

  den = (fit.uz2 * fit.u0 - fit.uz * fit.uz);
  if (fabs(den) < 10e-10)
    den = 1.f;
  t.ty = (fit.uyz * fit.u0  - fit.uy  * fit.uz) / den;
  t.y0 = (fit.uy  * fit.uz2 - fit.uyz * fit.uz) / den;
}

/** Simple implementation of the Kalman Filter selection on the GPU (step 4).

Will rely on pre-processing for selecting next-hits for each hit.

Implementation,
- Perform implementation searching on all hits for each sensor

The algorithm has two parts:
- Track creation (two hits)
- Track following (consecutive sensors)


Optimizations,
- Optimize with shared memory
- Optimize further with pre-processing

Then there must be a post-processing, which selects the
best tracks based on (as per the conversation with David):
- length
- chi2

For this, simply use the table with all created tracks (postProcess):

#track, h0, h1, h2, h3, ..., hn, length, chi2

*/

__global__ void gpuKalman(Track* tracks, bool* track_holders) {
  Track t;
  TrackFit tfit;
  Sensor s0, s1, s2;
  Hit h0, h1, h2;

  float fit, best_fit;
  bool fit_is_better, accept_track;
  int best_hit, best_hit_h2;

  // 4 of the sensors are unused, because the algorithm needs 5-sensor spans
  const int current_sensor = (gridDim.x - blockIdx.x - 1 + 4);

  s0.hitStart = sensor_hitStarts[current_sensor];
  s0.hitNums  = sensor_hitNums[current_sensor];
  s0.z        = sensor_Zs[current_sensor];

  const int next_sensor  = current_sensor - 2;
  const int third_sensor = current_sensor - 4;

  // TODO: shared memory.
  s1.hitStart = sensor_hitStarts[next_sensor];
  s1.hitNums  = sensor_hitNums[next_sensor];
  s1.z        = sensor_Zs[next_sensor];

  // Iterate over all hits for the current sensor
  const int hits_per_thread = (s0.hitNums + blockDim.x - 1) / blockDim.x;
  for (int i = 0; i < hits_per_thread; ++i) {
    const int current_hit = threadIdx.x * hits_per_thread + i;
    if (current_hit < s0.hitNums) {
      h0.x = hit_Xs[s0.hitStart + current_hit];
      h0.y = hit_Ys[s0.hitStart + current_hit];

      // Initialize track
      for (int j = 0; j < MAX_TRACK_SIZE; ++j)
        t.hits[j] = -1;

      // TRACK CREATION
      // TODO: Modify with preprocessed list of hits.
      best_fit    = MAX_FLOAT;
      best_hit    = -1;
      best_hit_h2 = -1;
      for (int j = 0; j < sensor_hitNums[next_sensor]; ++j) {
        // TODO: Load in chunks of SHARED_MEMORY and take
        // them from shared memory.
        h1.x = hit_Xs[s1.hitStart + j];
        h1.y = hit_Ys[s1.hitStart + j];

        // Search in both directions
        for (int displ = 1; displ > -3; --displ) {
          if (third_sensor + displ >= 0) {
            // TODO: shared memory.
            s2.hitStart = sensor_hitStarts[third_sensor + displ];
            s2.hitNums  = sensor_hitNums[third_sensor + displ];
            s2.z        = sensor_Zs[third_sensor + displ];

            // Iterate in the third! list of hits
            for (int k = 0; k < sensor_hitNums[third_sensor + displ]; ++k) {
              // TODO: Load in chunks of SHARED_MEMORY and take
              // them from shared memory.
              h2.x = hit_Xs[s2.hitStart + k];
              h2.y = hit_Ys[s2.hitStart + k];

              fit = fitHits(h0, h1, h2, s0, s1, s2);
              fit_is_better = fit < best_fit;

              best_fit    = fit_is_better * fit + !fit_is_better * best_fit;
              best_hit    = fit_is_better * j   + !fit_is_better * best_hit;
              best_hit_h2 = fit_is_better * k   + !fit_is_better * best_hit_h2;
            }
          }
        }
      }

      accept_track = best_fit != MAX_FLOAT;

      // We have a best fit!

      // For those that have tracks, we go on
      if (accept_track) {
        // Fill in t (ONLY in case the best fit is acceptable)
        acceptTrack(t, tfit, h0, h1, s0, s1, s0.hitStart + current_hit, s1.hitStart + best_hit);
        updateTrack(t, tfit, h2, s2, s2.hitStart + best_hit_h2);

        // TRACK FOLLOWING
        for (int follow_sensor = next_sensor - 4; follow_sensor >= 0; follow_sensor -= 2) {
          s1.hitStart = sensor_hitStarts[follow_sensor];
          s1.hitNums  = sensor_hitNums[follow_sensor];
          s1.z        = sensor_Zs[follow_sensor];

          best_fit = MAX_FLOAT;
          for (int k = 0; k < sensor_hitNums[follow_sensor]; ++k) {
            // TODO: Load in chunks of SHARED_MEMORY and take them from shared memory.
            h1.x = hit_Xs[s1.hitStart + k];
            h1.y = hit_Ys[s1.hitStart + k];

            fit = fitHitToTrack(t, h1, s1);
            fit_is_better = fit < best_fit;

            best_fit = fit_is_better * fit + !fit_is_better * best_fit;
            best_hit = fit_is_better * k   + !fit_is_better * best_hit;
          }

          // We have a best fit!
          // Fill in t, ONLY in case the best fit is acceptable

          // TODO: Maybe try to do this more "parallel"
          if (best_fit != MAX_FLOAT)
            updateTrack(t, tfit, h1, s1, s1.hitStart + best_hit);
        }
      }

      // If it's a track, write it to memory, no matter what kind
      // of track it is.
      track_holders[s0.hitStart + current_hit] = accept_track && (t.hitsNum >= MIN_HITS_TRACK);
      if (accept_track && (t.hitsNum >= MIN_HITS_TRACK))
        tracks[s0.hitStart + current_hit] = t;
    }
  }
}


/* Calculating the chi2 of a track is quite cumbersome.
It implies loading hit_Xs, hit_Ys, and sensor_Zs elements for each
hit of the track. This introduces branching, and is slow.

However, the track chi2 has to be calculated only when the
track has been created (the tx, ty values change).
*/

__device__ float trackChi2(Track& t) {
  float ch = 0.0;
  int nDoF  = -4 + 2 * t.hitsNum;
  Hit h;
  for (int i = 0; i < MAX_TRACK_SIZE; i++) {
    // TODO: Maybe there's a better way to do this
    if (t.hits[i] != -1) {
      h.x = hit_Xs[t.hits[i]];
      h.y = hit_Ys[t.hits[i]];

      ch += hitChi2(t, h, hit_Zs[t.hits[i]]);
    }
  }
  return ch/nDoF;
}

__device__ float hitChi2(Track& t, Hit& h, int hit_z) {
  // chi2 of a hit
  float dx = (t.x0 + t.tx * hit_z) - h.x;
  float dy = (t.y0 + t.ty * hit_z) - h.y;
  return dx * dx * PARAM_W + dy * dy * PARAM_W;
}


/** The postProcess method takes care of discarding redundant tracks. In other
 words, it (hopefully) increases the purity of our tracks.

- Inspect track_holders and generate track_indexes and num_tracks.

The main idea is to accept tracks which have unique (> REQUIRED_UNIQUES) hits.
For this, each track is checked against all other more preferent tracks, and non
common hits are kept.

TODO: Change the preference system into something more civilized.  A track t0
has preference over another t1 one if: t0.hitsNum > t1.hitsNum || (t0.hitsNum ==
t1.hitsNum && chi2(t0) < chi2(t1))
*/
__global__ void postProcess(Track* tracks, bool* track_holders, int* track_indexes, int* num_tracks, int* tracks_to_process) {
  // tracks_to_process holds the list of tracks with track_holders[t] == true

  // TODO: Try with sh_tracks_to_process
  // __shared__ int sh_tracks_to_process[MAX_POST_TRACKS];

  __shared__ Track sh_tracks[BUNCH_POST_TRACKS];
  __shared__ float sh_chi2[BUNCH_POST_TRACKS];

  __shared__ Track sh_next_tracks[BUNCH_POST_TRACKS];
  __shared__ float sh_next_chi2[BUNCH_POST_TRACKS];

  // We will use an atomic to write on a vector concurrently on several values
  __shared__ int tracks_to_process_size;
  __shared__ int tracks_accepted_size;

  tracks_to_process_size = 0;
  tracks_accepted_size = 0;

  __syncthreads(); // for the atomics tracks_to_process_size, and tracks_processed

  int current_track, next_track;
  bool preferent;

  for (int i = 0; i < (*no_hits + blockDim.x - 1) / blockDim.x; ++i) {
    current_track = blockDim.x * i + threadIdx.x;
    if (current_track < *no_hits) {
      // Iterate in all tracks (current_track)

      if (track_holders[current_track]) {
        // Atomic add
        int current_atomic = atomicAdd(&tracks_to_process_size, 1);

        // TODO: This condition shouldn't exist,
        // redo using method to process in batches if necessary
        // if (current_atomic < MAX_POST_TRACKS)
        tracks_to_process[current_atomic] = current_track;
      }
    }
  }

  __syncthreads();

  // Iterate in all current_tracks against all next_tracks.
  // Do this processing on batches of blockDim.x size
  for (int i = 0; i < (tracks_to_process_size + blockDim.x - 1) / blockDim.x; ++i) {
    current_track = blockDim.x * i + threadIdx.x;
    if (current_track < tracks_to_process_size) {
      // Store all tracks in sh_tracks
      sh_tracks[threadIdx.x] = tracks[tracks_to_process[current_track]];

      // Calculate chi2
      sh_chi2[threadIdx.x] = trackChi2(sh_tracks[threadIdx.x]);
    }

    __syncthreads();

    // Iterate in all next_tracks
    for (int j = 0; j < (tracks_to_process_size + blockDim.x - 1) / blockDim.x; ++j) {
      next_track = blockDim.x * j + threadIdx.x;

      if (next_track < tracks_to_process_size) {
        // Store all tracks in sh_tracks
        sh_next_tracks[threadIdx.x] = tracks[tracks_to_process[next_track]];

        // Calculate chi2
        sh_next_chi2[threadIdx.x] = trackChi2(sh_tracks[threadIdx.x]);
      }

      __syncthreads();

      // All is loaded, commencing assault!
      for (int k=0; k<BUNCH_POST_TRACKS; ++k) {
        next_track = blockDim.x * j + k;

        if (current_track < tracks_to_process_size && next_track < tracks_to_process_size) {
          /* Compare all tracks to check uniqueness, based on
          - length
          - chi2

          preferent is a boolean storing this logic. It reads,

          TODO: Change preference system by something more civilized
          next_track is preferent if
            it's not current_track,
            its length > current_track . length OR
            (its length == current_track . length AND
            chi2 < current_track . chi2)
          */
          preferent = current_track!=next_track &&
                    (sh_next_tracks[k].hitsNum > sh_tracks[threadIdx.x].hitsNum ||
                    (sh_next_tracks[k].hitsNum == sh_tracks[threadIdx.x].hitsNum &&
                    sh_next_chi2[k] < sh_chi2[threadIdx.x]));

          // Preference system based solely on chi2
          /*preferent = current_track!=next_track &&
                    sh_next_chi2[k] < sh_chi2[threadIdx.x]; */

          // TODO: Maybe there's a better way...
          if (preferent) {
            // Eliminate hits from current_track, based on next_track's
            for (int current_hit=0; current_hit<MAX_TRACK_SIZE; ++current_hit) {
              for (int next_hit=0; next_hit<MAX_TRACK_SIZE; ++next_hit) {
                /* apply mask:
                a[i] =
                  (a[i] == b[j]) * -1 +
                  (a[i] != b[j]) * a[i]
                */
                sh_tracks[threadIdx.x].hits[current_hit] =
                  (sh_tracks[threadIdx.x].hits[current_hit] == sh_next_tracks[k].hits[next_hit]) * -1 +
                  (sh_tracks[threadIdx.x].hits[current_hit] != sh_next_tracks[k].hits[next_hit]) *
                    sh_tracks[threadIdx.x].hits[current_hit];
              }
            }
          }
        }
      }
    }

    if (current_track < tracks_to_process_size) {
      // Check how many uniques do we have
      int unique = 0;
      for (int hit=0; hit<MAX_TRACK_SIZE; ++hit)
        unique += (sh_tracks[threadIdx.x].hits[hit]!=-1);

      if (!POST_PROCESSING || ((float) unique) / sh_tracks[threadIdx.x].hitsNum > REQUIRED_UNIQUES) {
        int current_track_accepted = atomicAdd(&tracks_accepted_size, 1);

        track_indexes[current_track_accepted] = tracks_to_process[current_track];
      }
    }
  }

  __syncthreads();

  if (threadIdx.x==0)
    *num_tracks = tracks_accepted_size;
}

