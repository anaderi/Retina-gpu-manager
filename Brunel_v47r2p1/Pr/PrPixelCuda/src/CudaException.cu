#include "CudaException.h"

#include <sstream>

using namespace std;

CudaException::CudaException(hipError_t err, const char * statement) :
    runtime_error(makeMessage(err, statement)) {
}

CudaException::~CudaException() throw() {
}

string CudaException::makeMessage(hipError_t err, const char * statement) {
  ostringstream str;
  str << "Failed to run: '" << statement << "'. " << hipGetErrorString(err);
  return str.str();
}
