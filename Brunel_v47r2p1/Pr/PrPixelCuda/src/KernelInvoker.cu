#include "Kernel.cuh"
#include "KernelInvoker.cuh"
#include "KernelPreprocessing.h"

#include <boost/timer/timer.hpp>

#include <iostream>
#include <utility>
#include <vector>

hipError_t invokeParallelSearch(
    dim3                         numThreads,
    const std::vector<uint8_t> & input,
    std::vector<uint8_t>       & solution,
    std::ostream               & logger) {
  boost::timer::auto_cpu_timer t;

  // For now, just perform what we did before
  // (backwards compatibility)
  logger << "Input pointer: "
    << std::hex << "0x" << (long long int) &(input[0])
    << std::dec << std::endl;

  Event event;
  parseEvent(const_cast<uint8_t*>(&input[0]), input.size(), event);
  printInfo(event, logger);

  char  *dev_input             = 0;
  int   *dev_num_tracks        = 0;
  int   *dev_track_indexes     = 0;
  Track *dev_tracks            = 0;
  bool  *dev_track_holders     = 0;
  int   *dev_prevs             = 0;
  int   *dev_nexts             = 0;
  int   *dev_tracks_to_process = 0;
  hipError_t cudaStatus = hipSuccess;

  // Choose which GPU to run on, change this on a multi-GPU system.
  cudaCheck(hipSetDevice(0));

  // Allocate memory
  // Allocate CPU buffers
  Track *tracks          = (Track*) malloc(MAX_TRACKS * sizeof(Track));
  int   *num_tracks      = (int*)   malloc(sizeof(int));
  int   *h_prevs         = (int*)   malloc(event.no_hits[0] * sizeof(int));
  int   *h_nexts         = (int*)   malloc(event.no_hits[0] * sizeof(int));
  bool  *h_track_holders = (bool*)  malloc(MAX_TRACKS * sizeof(bool));
  int   *track_indices   = (int*)   malloc(MAX_TRACKS * sizeof(int));

  // Allocate GPU buffers
  cudaCheck(hipMalloc((void**)&dev_tracks,            MAX_TRACKS * sizeof(Track)));
  cudaCheck(hipMalloc((void**)&dev_track_holders,     MAX_TRACKS * sizeof(bool)));
  cudaCheck(hipMalloc((void**)&dev_track_indexes,     MAX_TRACKS * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_tracks_to_process, MAX_TRACKS * sizeof(int)));

  cudaCheck(hipMalloc((void**)&dev_prevs, event.no_hits[0] * sizeof(int)));
  cudaCheck(hipMalloc((void**)&dev_nexts, event.no_hits[0] * sizeof(int)));

  // Copy input file from host memory to GPU buffers
  cudaCheck(hipMalloc((void**)&dev_input,      input.size()));
  cudaCheck(hipMalloc((void**)&dev_num_tracks, sizeof(int)));

  // memcpys
  cudaCheck(hipMemcpy(dev_input, input.data(), input.size(), hipMemcpyHostToDevice));

  // clear track_holders flags
  cudaCheck(hipMemset(dev_track_holders, 0, MAX_TRACKS * sizeof(bool)));

  // Launch a kernel on the GPU with one thread for each element.
  prepareData<<<1, 1>>>(dev_input, dev_prevs, dev_nexts);

  //----------------------

  const int threadCount         = 32;
  const int candidatesPerThread = 100000;
  std::vector<Span> spans;
  splitHits(event, threadCount, candidatesPerThread, spans);

  Span * devSpans;
  const int devSpansSize = spans.size() * sizeof(Span);
  cudaCheck(hipMalloc((void**)&devSpans, devSpansSize));
  cudaCheck(hipMemcpy(devSpans, spans.data(), devSpansSize, hipMemcpyHostToDevice));

  const int numBlocks = spans.size() / threadCount;

  Fit * devFittings;
  cudaCheck(hipMalloc((void**)&devFittings, spans.size() * sizeof(Fit)));

  gpuKalmanBalanced<<<numBlocks, threadCount>>>(dev_tracks, devSpans, devFittings);
  cudaCheckLast("gpuKalmanBalanced");

  consolidateHits<<<1, 1>>>(devFittings, spans.size(), dev_tracks, dev_track_holders);
  cudaCheckLast("consolidateHits");

  // Launch a kernel on the GPU with one thread for each element.
  //prepareData<<<1, 1>>>(dev_input, dev_prevs, dev_nexts, dev_track_holders);

  // gpuKalman
  logger << "gpuKalman" << std::endl;
  //hipEvent_t start_kalman, start_postprocess, stop;
  //float t0, t1, t2;

  //hipEventCreate(&start_kalman);
  //hipEventCreate(&start_postprocess);
  //hipEventCreate(&stop);

  //hipEventRecord(start_kalman, 0);

  // 4 of the sensors are unused, because the algorithm needs 5-sensor spans
  //const int effective_no_sensors = *event.no_sensors - 4;
  //gpuKalman<<<effective_no_sensors, numThreads>>>(dev_tracks, dev_track_holders);

  //hipEventRecord(start_postprocess);

  logger << "postProcess" << std::endl;
  postProcess<<<1, numThreads>>>(dev_tracks, dev_track_holders, dev_track_indexes, dev_num_tracks, dev_tracks_to_process);

  cudaCheck(hipDeviceSynchronize());

  //hipEventRecord(stop, 0);
  //hipEventSynchronize(stop);

  //hipEventElapsedTime(&t0, start_kalman, start_postprocess);
  //hipEventElapsedTime(&t1, start_postprocess, stop);
  //hipEventElapsedTime(&t2, start_kalman, stop);
  //hipEventDestroy(start_kalman);
  //hipEventDestroy(start_postprocess);
  //hipEventDestroy(stop);

  // get the results
  cudaCheck(hipMemcpy(h_track_holders, dev_track_holders, MAX_TRACKS * sizeof(bool), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(track_indices, dev_track_indexes, MAX_TRACKS * sizeof(int), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(tracks, dev_tracks, MAX_TRACKS * sizeof(Track), hipMemcpyDeviceToHost));
  cudaCheck(hipMemcpy(num_tracks, dev_num_tracks, sizeof(int), hipMemcpyDeviceToHost));

  // number of tracks after stage#1
  int no_tracks_stage1 = 0;
  for (int i = 0; i < event.no_hits[0]; ++i) {
    if (h_track_holders[i])
      ++no_tracks_stage1;
  }

  // copy selected track to the solution vector
  if (*num_tracks > 0) {
    solution.resize(*num_tracks * sizeof(Track));
    Track * solutionTracks = (Track*)&solution[0];
    for (size_t i = 0; i != *num_tracks; ++i)
      solutionTracks[i] = tracks[track_indices[i]];
  }

  // print debug info
  for (int i = 0; i < num_tracks[0]; ++i)
    printTrack(tracks, track_indices[i], event, logger);
  logger << "Processed " << num_tracks[0] << " tracks" << std::endl;

  free(h_prevs);
  free(h_nexts);
  free(h_track_holders);
  free(tracks);
  free(num_tracks);

  cudaCheck(hipFree(devSpans));
  cudaCheck(hipFree(devFittings));

  return cudaStatus;
}

// #track, h0, h1, h2, h3, ..., hn, length, chi2
void printTrack(Track* tracks, int track_no, const Event & event, std::ostream& logger) {
  logger << track_no << ": ";

  Track t = tracks[track_no];
  for (int i = 0; i < t.hitsNum; ++i)
    logger << event.hit_IDs[t.hits[i]] << ", ";

  logger << "length: " << (int) t.hitsNum << std::endl;
}

void printOutAllSensorHits(int* prevs, int* nexts, const Event & event, std::ostream& logger) {
  logger << "All valid sensor hits: " << std::endl;
  for (int i = 0; i < *event.no_sensors; ++i) {
    for (int j = 0; j < event.sensor_hitNums[i]; ++j) {
      int hit = event.sensor_hitStarts[i] + j;

      if (nexts[hit] != -1)
        logger << hit << ", " << nexts[hit] << std::endl;
    }
  }
}

void printOutSensorHits(int sensorNumber, int* prevs, int* nexts, const Event & event, std::ostream& logger) {
  for (int i = 0; i < event.sensor_hitNums[sensorNumber]; ++i) {
    int hstart = event.sensor_hitStarts[sensorNumber];

    logger << hstart + i << ": " << prevs[hstart + i] << ", " << nexts[hstart + i] << std::endl;
  }
}

void printInfo(const Event & event, std::ostream& logger) {
  logger << "Read info:" << std::endl
    << " no sensors: " << event.no_sensors << std::endl
    << " no hits: " << event.no_hits << std::endl
    << "First 5 sensors: " << std::endl;

  for (int i = 0; i < 5; ++i) {
    logger << " Zs: " << event.sensor_Zs[i] << std::endl
      << " hitStarts: " << event.sensor_hitStarts[i] << std::endl
      << " hitNums: " << event.sensor_hitNums[i] << std::endl << std::endl;
  }

  logger << "First 5 hits: " << std::endl;

  for (int i = 0; i < 5; ++i) {
    logger << " hit_id: " << event.hit_IDs[i] << std::endl;
    logger << " hit_X:  " << event.hit_Xs[i]  << std::endl;
    logger << " hit_Y:  " << event.hit_Ys[i]  << std::endl;
    logger << " hit_Z:  " << event.hit_Zs[i]  << std::endl;
    logger << std::endl;
  }
}

void getMaxNumberOfHits(char*& input, int& maxHits) {
  int * l_no_sensors       = (int*) input;
  int * l_no_hits          = (int*) (l_no_sensors + 1);
  int * l_sensor_Zs        = (int*) (l_no_hits + 1);
  int * l_sensor_hitStarts = (int*) (l_sensor_Zs + l_no_sensors[0]);
  int * l_sensor_hitNums   = (int*) (l_sensor_hitStarts + l_no_sensors[0]);

  maxHits = 0;
  for (int i = 0; i < l_no_sensors[0]; ++i) {
    if (l_sensor_hitNums[i] > maxHits)
      maxHits = l_sensor_hitNums[i];
  }
}
